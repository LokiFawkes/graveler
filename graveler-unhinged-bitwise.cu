
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <stdio.h>

#define ROUNDS 1000000000
#define ROLLS 231
#define BLOCKSIZE 1024

/*
*
	This is made to simulate the number of times a 1 is rolled on a d4 per round of 231 rolls, similar to Austin Hourigan's graveler.py
	I've taken some liberties to optimize it further, in addition to making it run in CUDA.
	The kernel, __global__ void sim_rolls, does the heavy lifting, as it runs on the CUDA cores, reporting back to the system after completing.
	Unnecessary arrays were dropped, instead keeping track of the fact that the rolls were done and how many times in a round a one was rolled, AKA graveler was too paralyzed to move.
	The unnecessary check for 177 ones has been dropped. This is running too fast to stop at a precise target.
	In this version, the number of roll sessions can overflow, but should not go below target. This wastes valuable time on excess roll sessions, but is faster than my other code because it's not constructing and destructing threads over and over.
	Two kills are included - One within the stream that runs the simulation, one in a dedicated stream for stopping the simulation.
	Now even more unhinged, trying out bitwise optimization. 
*
*/

__global__ void sim_stop(unsigned long long *d_rolls, bool *d_kill, int *d_blocks) {
    while(*d_rolls < ROUNDS - (*d_blocks * BLOCKSIZE)) {
        if(*d_rolls >= ROUNDS - *d_blocks) *d_kill = true; return;
    }
    *d_kill = true;
    return;
}

__global__ void sim_rolls(int *d_maxOnes, unsigned long long *d_rolls, int seed, bool *d_kill) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= ROUNDS) return;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    while(*d_rolls < ROUNDS - (idx * 2) && !*d_kill){
        int ones = 0;
        int curRolls = 0;
        for (int i = 0; i < 16; i++){
            unsigned int roll = hiprand(&state);
            for (int j = 0; j < 16; j++){
                if(curRolls < 231){
                    int shift = (j * 2);
                    unsigned int currentRoll = (roll >> shift) & 0x03;
                    if(currentRoll == 0) ones++;
                    curRolls++;
                }
            }
        }
        if(*d_kill) return;
        // atomicMax writes the number of ones rolled this session to d_maxOnes if it is greater, atomicAdd increments d_rolls by one each session
        atomicMax(d_maxOnes, ones);
        atomicAdd(d_rolls, 1);
        
    }
}

int main() {
    int maxOnes = 0;
    int *d_maxOnes;
    unsigned long long rolls = 0;
    unsigned long long *d_rolls;
    bool kill = false;
    bool *d_kill;
    hipDeviceProp_t prop;
    int deviceId;
    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&prop, deviceId);
    int smCount = prop.multiProcessorCount;
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1); hipStreamCreate(&stream2);
    // Allocate memory for CUDA copies of maxOnes and rolls
    hipMalloc(&d_maxOnes, sizeof(int));
    hipMalloc(&d_rolls, sizeof(unsigned long long));
    hipMalloc(&d_kill, sizeof(bool));

    // Initialize these values to 0 on the card
    hipMemcpy(d_maxOnes, &maxOnes, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rolls, &rolls, sizeof(unsigned long long), hipMemcpyHostToDevice);
    hipMemcpy(d_kill, &kill, sizeof(bool), hipMemcpyHostToDevice);

    // Set block size, allocate blocks, run kernel
    int blockSize = BLOCKSIZE;
    // The code now polls the CUDA device for how many blocks it can run. The kernel will kill itself thread by thread when the target number of sessions has been reached.
    int maxActiveBlocks = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, sim_rolls, blockSize, 0);
    int numBlocks = smCount * maxActiveBlocks;
    // Passing number of blocks to device for the dedicated killswitch kernel
    int *d_blocks;
    hipMalloc(&d_blocks, sizeof(int));
    hipMemcpy(d_blocks, &numBlocks, sizeof(int), hipMemcpyHostToDevice);
    //New timing method using cuda events
    float totalTime=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    sim_stop<<<1, 1, 0, stream1>>>(d_rolls, d_kill, d_blocks);
    sim_rolls<<<numBlocks, blockSize, 0, stream2>>>(d_maxOnes, d_rolls, time(NULL), d_kill);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&totalTime, start, stop);
    
    // Copy the result back to system memory, now that the CUDA program is over
    hipMemcpy(&maxOnes, d_maxOnes, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&rolls, d_rolls, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Report the important numbers
    std::cout << "Highest Ones Roll: " << maxOnes << std::endl;
    std::cout << "Number of Roll Sessions: " << rolls << std::endl;
    std::cout << totalTime << "ms" << std::endl;
    // Never malloc without a free
    hipFree(d_maxOnes);
    hipFree(d_rolls);
    hipFree(d_kill);
    hipFree(d_blocks);
    return 0;
}
